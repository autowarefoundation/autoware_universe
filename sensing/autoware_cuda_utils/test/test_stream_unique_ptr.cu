// Copyright 2025 Tier IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "autoware/cuda_utils/cuda_check_error.hpp"

#include <autoware/cuda_utils/stream_unique_ptr.hpp>

#include <gtest/gtest.h>

TEST(StreamUniquePtrTest, MakeCudaStreamDefault)
{
  // Test creating a CUDA stream with default flags
  auto stream = autoware::cuda_utils::makeCudaStream();
  EXPECT_NE(stream.get(), nullptr);

  // Check that the stream is valid
  hipStream_t raw_stream = *stream.get();
  EXPECT_EQ(hipStreamQuery(raw_stream), hipSuccess);
}

TEST(StreamUniquePtrTest, MakeCudaStreamWithFlags)
{
  // Test creating a CUDA stream with custom flags
  auto stream = autoware::cuda_utils::makeCudaStream(hipStreamNonBlocking);
  EXPECT_NE(stream.get(), nullptr);

  // Check that the stream is valid
  hipStream_t raw_stream = *stream.get();
  EXPECT_EQ(hipStreamQuery(raw_stream), hipSuccess);

  // Check that the stream is non-blocking
  unsigned int flags{};
  CHECK_CUDA_ERROR(hipStreamGetFlags(raw_stream, &flags));
  EXPECT_EQ(flags, hipStreamNonBlocking);
}

TEST(StreamUniquePtrTest, StreamDeleterFunctionality)
{
  // Test that StreamDeleter properly handles cleanup
  {
    auto stream = autoware::cuda_utils::makeCudaStream();
    // Deleter will be called automatically on scope exit
  }
}

TEST(StreamUniquePtrTest, StreamReset)
{
  // Test that we can reset a stream
  auto stream = autoware::cuda_utils::makeCudaStream();
  EXPECT_NE(stream.get(), nullptr);
  stream.reset();
  EXPECT_EQ(stream.get(), nullptr);
}

TEST(StreamUniquePtrTest, StreamMove)
{
  // Test moving a stream unique pointer
  auto stream1 = autoware::cuda_utils::makeCudaStream();
  EXPECT_NE(stream1.get(), nullptr);

  auto stream2 = std::move(stream1);
  EXPECT_EQ(stream1.get(), nullptr);
  EXPECT_NE(stream2.get(), nullptr);

  // Check that the moved stream is still valid
  EXPECT_EQ(hipStreamQuery(*stream2), hipSuccess);
}
