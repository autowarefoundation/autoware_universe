// Copyright 2025 Tier IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "autoware/cuda_utils/cuda_check_error.hpp"
#include "autoware/cuda_utils/cuda_unique_ptr.hpp"

#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

TEST(CudaUniquePtrTest, MakeUniqueDeviceMemory)
{
  // Test creating a single object on device
  auto ptr = autoware::cuda_utils::make_unique<float>();
  EXPECT_NE(ptr.get(), nullptr);
}

TEST(CudaUniquePtrTest, MakeUniqueDeviceArray)
{
  // Test creating an array on device
  auto ptr = autoware::cuda_utils::make_unique<float[]>(100);
  EXPECT_NE(ptr.get(), nullptr);

  hipPointerAttribute_t attributes{};
  CHECK_CUDA_ERROR(hipPointerGetAttributes(&attributes, ptr.get()));
  EXPECT_EQ(attributes.type, hipMemoryTypeDevice);
  EXPECT_EQ(attributes.devicePointer, ptr.get());
}

TEST(CudaUniquePtrTest, MakeUniqueHostMemory)
{
  // Test creating a single object on host
  auto ptr = autoware::cuda_utils::make_unique_host<float>();
  EXPECT_NE(ptr.get(), nullptr);

  hipPointerAttribute_t attributes{};
  CHECK_CUDA_ERROR(hipPointerGetAttributes(&attributes, ptr.get()));
  EXPECT_EQ(attributes.type, hipMemoryTypeHost);
  EXPECT_EQ(attributes.hostPointer, ptr.get());
}

TEST(CudaUniquePtrTest, MakeUniqueHostArray)
{
  // Test creating an array on host
  auto ptr = autoware::cuda_utils::make_unique_host<float[]>(100, hipHostMallocDefault);
  EXPECT_NE(ptr.get(), nullptr);
}

TEST(CudaUniquePtrTest, DeleterFunctionality)
{
  // Test that CudaDeleter and CudaDeleterHost types exist and are usable
  {
    auto ptr = autoware::cuda_utils::make_unique<int>();
    // Deleter will be called automatically on scope exit
  }

  {
    auto ptr = autoware::cuda_utils::make_unique_host<int>();
    // Deleter will be called automatically on scope exit
  }

  // If we reach here without crashes, deleters worked correctly
  SUCCEED();
}
